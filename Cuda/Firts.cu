#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#if defined(NDEBUG)
#define CUDA_CHECK(X) (X)
#else
#define CUDA_CHECK(X) do{\
	(X);\
	hipError_t e = hipGetLastError();\
	if(hipSuccess != e){\
		printf("cuda failure %s at %s : %d",hipGetErrorString(e), __FILE__, __LINE__);\
		exit(1);\
	}\
}while(0)
#endif



__global__ void mulKernel(int* c, const int* a, const int* b, const int WIDTH) {
	int x = threadIdx.x;
	int y = threadIdx.y;
	int i = y * WIDTH + x;

	int sum = 0;
	for (int k = 0; k < WIDTH; k++) {
		sum += a[y * WIDTH + k] * b[k * WIDTH + x];
	}
	c[i] = sum;
}

int main()
{
	const int WIDTH = 5;
	int a[WIDTH][WIDTH];
	int b[WIDTH][WIDTH];
	int c[WIDTH][WIDTH] = { 0 };

	for (int y = 0; y < WIDTH; y++) {
		for (int x = 0; x < WIDTH; x++) {
			a[y][x] = rand() % 20;
			b[y][x] = rand() % 20;
		}
	}

    //imprimiendo matriz A
    printf("VALORES DE MATRIZ A \n");
	for (int y = 0; y < WIDTH; y++)
	{
		for (int x = 0; x < WIDTH; x++)
		{
			printf("%5d", a[y][x]);
		}
		printf("\n");
	}
	
	printf("\n\n VALORES DE MATRIZ A \n");
	for (int y = 0; y < WIDTH; y++)
	{
		for (int x = 0; x < WIDTH; x++)
		{
			printf("%5d", b[y][x]);
		}
		printf("\n\n");
	}
	
	//device side data
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;

	//allocate device memory
	hipMalloc((void**)&dev_a, WIDTH * WIDTH * sizeof(int));
	hipMalloc((void**)&dev_b, WIDTH * WIDTH * sizeof(int));
	hipMalloc((void**)&dev_c, WIDTH * WIDTH * sizeof(int));

	//copy from host to device
	hipMemcpy(dev_a, a, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice);	//dev_a = a
	hipMemcpy(dev_b, b, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice);	//dev_b = b

	//launch a kernel on the GPU with one thread for each element
	dim3 dimBlock(WIDTH, WIDTH, 1);	//x,y,z
	mulKernel << <1, dimBlock >> > (dev_c, dev_a, dev_b, WIDTH);
	CUDA_CHECK(hipPeekAtLastError());

	//copy from device to host
	hipMemcpy(c, dev_c, WIDTH * WIDTH * sizeof(int), hipMemcpyDeviceToHost);

	//free device memory
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	//print the result
	printf("RESULTADO DE MULTIPLICACION \n");
	for (int y = 0; y < WIDTH; y++) {
		for (int x = 0; x < WIDTH; x++) {
			printf("%10d", c[y][x]);
		}
		printf("\n");
	}
	return 0;
}