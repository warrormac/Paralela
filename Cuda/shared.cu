#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#if defined(NDEBUG)
#define CUDA_CHECK(X) (X)
#else
#define CUDA_CHECK(X) do{\
	(X);\
	hipError_t e = hipGetLastError();\
	if(hipSuccess != e){\
		printf("cuda failure %s at %s : %d",hipGetErrorString(e), __FILE__, __LINE__);\
		exit(1);\
	}\
}while(0)
#endif
#define Tile_size 16

						
__global__ void MatrixMulKernel(int* d_M, int* d_N, int* d_P, int Width) {
    
    __shared__ int Mds[Tile_size][Tile_size]; 
    __shared__ int Nds[Tile_size][Tile_size];
				
    int bx = blockIdx.x; 
    int by = blockIdx.y; 
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    						
    // Identify the row and column of the d_P element to work on 
    int Row = by * Tile_size + ty;
    int Col = bx * Tile_size + tx;
    
    
    int Pvalue = 0;
    // Loop over the d_M and d_N tiles required to compute d_P element 
    for (int ph = 0; ph < Width/Tile_size; ++ph) {
    				
        // Collaborative loading of d_M and d_N tiles into shared memory 
        Mds[ty][tx] = d_M[Row*Width + ph*Tile_size + tx];
        Nds[ty][tx] = d_N[(ph*Tile_size + ty)*Width + Col]; 
        __syncthreads();
        						
        for (int k = 0; k < Tile_size; ++k) { 
            Pvalue += Mds[ty][k] * Nds[k][tx];
        						
        }
        __syncthreads();
        						
    }

    d_P[Row*Width + Col] = Pvalue; 
}

int main()
{
	int WIDTH;
    printf("%5d Tile_size ", Tile_size);
    printf("\n INGRESAR TAMAÑO DE LAS MATRICES:");
    scanf("%d",&WIDTH);
    
	int a[WIDTH][WIDTH];
	int b[WIDTH][WIDTH];
	int c[WIDTH][WIDTH] = { 1 };

	for (int y = 0; y < WIDTH; y++) {
		for (int x = 0; x < WIDTH; x++) {
			a[y][x] = rand() % 20;
			b[y][x] = rand() % 20;
		}
	}
	
	//device side data
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;

	//allocate device memory
	hipMalloc((void**)&dev_a, WIDTH * WIDTH * sizeof(int));
	hipMalloc((void**)&dev_b, WIDTH * WIDTH * sizeof(int));
	hipMalloc((void**)&dev_c, WIDTH * WIDTH * sizeof(int));
    
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

	//copy from host to device
	hipMemcpy(dev_a, a, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice);	//dev_a = a
	hipMemcpy(dev_b, b, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice);	//dev_b = b

	//launch a kernel on the GPU with one thread for each element
	dim3 dimGrid((WIDTH/Tile_size) + 1, (WIDTH/Tile_size) + 1, 1);//Number of Blocks required
    dim3 dimBlock(Tile_size, Tile_size, 1);//Number of threads in each block
    
    hipEventRecord(start);
	MatrixMulKernel << <dimGrid, dimBlock >> > (dev_a, dev_b, dev_c, WIDTH);
	hipEventRecord(stop);
	CUDA_CHECK(hipPeekAtLastError());

	//copy from device to host
	hipMemcpy(c, dev_c, WIDTH * WIDTH * sizeof(int), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
	//free device memory
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	
	printf("%fn <-TIME ", milliseconds);
	return 0;
}

